/* File: fd2d_3_2.cu
 * Name: D.Saravanan
 * Date: 18/01/2022
 * Simulation of a propagating sinusoidal in free space in the transverse
 * magnetic (TM) mode with the two-dimensional perfectly matched layer (PML)
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define idx (blockIdx.x * blockDim.x + threadIdx.x)
#define idy (blockIdx.y * blockDim.y + threadIdx.y)
#define stx (blockDim.x * gridDim.x)
#define sty (blockDim.y * gridDim.y)


typedef struct {
    float *fx1;
    float *fx2;
    float *fx3;
    float *fy1;
    float *fy2;
    float *fy3;
    float *gx2;
    float *gx3;
    float *gy2;
    float *gy3;
} pmlayer;


__device__
float sinusoidal(int t, float ds, float freq) {
    float dt = ds/6e8;  /* time step (s) */
    return sin(2 * M_PI * freq * dt * t);
}


__global__
void pmlparam(int npml, int nx, int ny, pmlayer *pml) {
    /* calculate the two-dimensional perfectly matched layer (PML) parameters */
    for (int n = threadIdx.x; n < npml; n += blockDim.x) {
        float xm = 0.33 * (npml-n)/npml*(npml-n)/npml*(npml-n)/npml;
        float xn = 0.33 * (npml-n-0.5)/npml*(npml-n-0.5)/npml*(npml-n-0.5)/npml;
        pml->fx1[n] = pml->fx1[nx-2-n] = pml->fy1[n] = pml->fy1[ny-2-n] = xn;
        pml->fx2[n] = pml->fx2[nx-2-n] = pml->fy2[n] = pml->fy2[ny-2-n] = 1/(1+xn);
        pml->gx2[n] = pml->gx2[nx-1-n] = pml->gy2[n] = pml->gy2[ny-1-n] = 1/(1+xm);
        pml->fx3[n] = pml->fx3[nx-2-n] = pml->fy3[n] = pml->fy3[ny-2-n] = (1-xn)/(1+xn);
        pml->gx3[n] = pml->gx3[nx-1-n] = pml->gy3[n] = pml->gy3[ny-1-n] = (1-xm)/(1+xm);
    }
}


__global__
void dfield(int t, int nx, int ny, pmlayer *pml, float *dz, float *hx, float *hy) {
    /* calculate the electric flux density Dz */
    for (int i = idy + 1; i < nx; i += sty) {
        for (int j = idx + 1; j < ny; j += stx) {
            int n = i*ny+j;
            dz[n] = pml->gx3[i] * pml->gy3[j] * dz[n] + pml->gx2[i] * pml->gy2[j] * 0.5 * (hy[n] - hy[n-ny] - hx[n] + hx[n-1]);
        }
    }
    __syncthreads();
    /* put a sinusoidal source at a point that is offset five cells
     * from the center of the problem space in each direction */
    if (idy == nx/2-5 && idx == ny/2-5)
        dz[(nx/2-5)*ny+(ny/2-5)] = sinusoidal(t, 0.01, 1500e6);
}


__global__
void efield(int nx, int ny, float *naz, float *dz, float *ez) {
    /* calculate the Ez field from Dz */
    for (int i = idy; i < nx; i += sty) {
        for (int j = idx; j < ny; j += stx) {
            int n = i*ny+j;
            ez[n] = naz[n] * dz[n];
        }
    }
}


__global__
void hfield(int nx, int ny, pmlayer *pml, float *ez, float *ihx, float *ihy, float *hx, float *hy) {
    /* calculate the Hx and Hy field */
    for (int i = idy; i < nx - 1; i += sty) {
        for (int j = idx; j < ny - 1; j += stx) {
            int n = i*ny+j;
            ihx[n] += ez[n] - ez[n+1];
            ihy[n] += ez[n] - ez[n+ny];
            hx[n] = pml->fy3[j] * hx[n] + pml->fy2[j] * (0.5 * ez[n] - 0.5 * ez[n+1] + pml->fx1[i] * ihx[n]);
            hy[n] = pml->fx3[i] * hy[n] - pml->fx2[i] * (0.5 * ez[n] - 0.5 * ez[n+ny] + pml->fy1[j] * ihy[n]);
        }
    }
}


int main() {

    int nx = 60;  /* number of grid points */
    int ny = 60;  /* number of grid points */

    int ns = 100;  /* number of time steps */

    float *dz, *ez, *hx, *hy;
    /* allocate unified memory accessible from host or device */
    hipMallocManaged(&dz, nx*ny*sizeof(*dz));
    hipMallocManaged(&ez, nx*ny*sizeof(*ez));
    hipMallocManaged(&hx, nx*ny*sizeof(*hx));
    hipMallocManaged(&hy, nx*ny*sizeof(*hy));

    /* initialize dz, ez, hx and hy arrays on the host */
    for (int i = 0; i < nx*ny; i++) {
        dz[i] = 0.0f;
        ez[i] = 0.0f;
        hx[i] = 0.0f;
        hy[i] = 0.0f;
    }

    float *ihx, *ihy;
    /* allocate unified memory accessible from host or device */
    hipMallocManaged(&ihx, nx*ny*sizeof(*ihx));
    hipMallocManaged(&ihy, nx*ny*sizeof(*ihy));

    /* initialize ihx and ihy arrays on the host */
    for (int i = 0; i < nx*ny; i++) {
        ihx[i] = 0.0f;
        ihy[i] = 0.0f;
    }

    float *naz;
    hipMallocManaged(&naz, nx*ny*sizeof(*naz));
    for (int i = 0; i < nx*ny; naz[i] = 1.0f, i++);

    float ds = 0.01;  /* spatial step (m) */
    float dt = ds/6e8;  /* time step (s) */

    pmlayer pml;
    hipMallocManaged(&pml.fx1, nx*sizeof(*pml.fx1));
    hipMallocManaged(&pml.fx2, nx*sizeof(*pml.fx2));
    hipMallocManaged(&pml.fx3, nx*sizeof(*pml.fx3));
    hipMallocManaged(&pml.fy1, ny*sizeof(*pml.fy1));
    hipMallocManaged(&pml.fy2, ny*sizeof(*pml.fy2));
    hipMallocManaged(&pml.fy3, ny*sizeof(*pml.fy3));
    hipMallocManaged(&pml.gx2, nx*sizeof(*pml.gx2));
    hipMallocManaged(&pml.gx3, nx*sizeof(*pml.gx3));
    hipMallocManaged(&pml.gy2, ny*sizeof(*pml.gy2));
    hipMallocManaged(&pml.gy3, ny*sizeof(*pml.gy3));

    for (int i = 0; i < nx; i++) {
        pml.fx1[i] = 0.0f;
        pml.fx2[i] = 1.0f;
        pml.fx3[i] = 1.0f;
        pml.gx2[i] = 1.0f;
        pml.gx3[i] = 1.0f;
    }

    for (int i = 0; i < ny; i++) {
        pml.fy1[i] = 0.0f;
        pml.fy2[i] = 1.0f;
        pml.fy3[i] = 1.0f;
        pml.gy2[i] = 1.0f;
        pml.gy3[i] = 1.0f;
    }

    dim3 gridDim, blockDim;
    blockDim.x = 16;
    blockDim.y = 16;
    gridDim.x = (ny + blockDim.x - 1)/blockDim.x;
    gridDim.y = (nx + blockDim.y - 1)/blockDim.y;

    int npml = 8;  /* pml thickness */
    pmlparam<<<gridDim, blockDim>>>(npml, nx, ny, &pml);

    for (int t = 1; t <= ns; t++) {
        dfield<<<gridDim, blockDim>>>(t, nx, ny, &pml, dz, hx, hy);
        efield<<<gridDim, blockDim>>>(nx, ny, naz, dz, ez);
        hfield<<<gridDim, blockDim>>>(nx, ny, &pml, ez, ihx, ihy, hx, hy);
    }

    hipDeviceSynchronize();

    hipFree(pml.fx1);
    hipFree(pml.fx2);
    hipFree(pml.fx3);
    hipFree(pml.fy1);
    hipFree(pml.fy2);
    hipFree(pml.fy3);
    hipFree(pml.gx2);
    hipFree(pml.gx3);
    hipFree(pml.gy2);
    hipFree(pml.gy3);
    hipFree(naz);
    hipFree(ihx);
    hipFree(ihy);
    hipFree(dz);
    hipFree(ez);
    hipFree(hx);
    hipFree(hy);

    return 0;
}
