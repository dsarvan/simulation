/* File: fd2d_3_3.cu
 * Name: D.Saravanan
 * Date: 19/01/2022
 * Simulation of a plane wave pulse propagating in free space in the transverse
 * magnetic (TM) mode with the two-dimensional perfectly matched layer (PML)
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define idx blockIdx.x*blockDim.x+threadIdx.x
#define idy blockIdx.y*blockDim.y+threadIdx.y
#define stx blockDim.x*gridDim.x
#define sty blockDim.y*gridDim.y


typedef struct {
    float *fx1;
    float *fx2;
    float *fx3;
    float *fy1;
    float *fy2;
    float *fy3;
    float *gx2;
    float *gx3;
    float *gy2;
    float *gy3;
} pmlayer;


__device__
float gaussian(int t, int t0, float sigma) {
    return exp(-0.5*(t - t0)/sigma*(t - t0)/sigma);
}


__global__
void ezinct(int ny, float *ezi, float *hxi, float *bc) {
    /* calculate the incident Ez */
    for (int j = idx+1; j < ny; j += stx) {
        ezi[j] += 0.5 * (hxi[j-1] - hxi[j]);
    }
    /* absorbing boundary conditions */
    if (idx == 0) ezi[0] = bc[0], bc[0] = bc[1], bc[1] = ezi[1];
    if (idx == ny-1) ezi[ny-1] = bc[3], bc[3] = bc[2], bc[2] = ezi[ny-2];
}


__global__
void dfield(int t, int nx, int ny, pmlayer *pml, float *ezi, float *dz, float *hx, float *hy) {
    /* calculate the electric flux density Dz */
    for (int i = idy+1; i < nx; i += sty) {
        for (int j = idx+1; j < ny; j += stx) {
            int n = i*ny+j;
            dz[n] = pml->gx3[i] * pml->gy3[j] * dz[n] + pml->gx2[i] * pml->gy2[j] * 0.5 * (hy[n] - hy[n-ny] - hx[n] + hx[n-1]);
        }
    }
    /* put a Gaussian pulse at the low end */
    if (idy == 0 && idx == 3) ezi[3] = gaussian(t, 20, 8.0f);
}


__global__
void inctdz(int nx, int ny, int npml, float *hxi, float *dz) {
    /* incident Dz values */
    for (int i = idx+npml-1; i <= nx-npml; i += stx) {
        dz[i*ny+(npml-1)] += 0.5 * hxi[npml-2];
        dz[i*ny+(ny-npml)] -= 0.5 * hxi[ny-npml];
    }
}


__global__
void efield(int nx, int ny, float *naz, float *dz, float *ez) {
    /* calculate the Ez field from Dz */
    for (int i = idy; i < nx; i += sty) {
        for (int j = idx; j < ny; j += stx) {
            int n = i*ny+j;
            ez[n] = naz[n] * dz[n];
        }
    }
}


__global__
void hxinct(int ny, float *ezi, float *hxi) {
    /* calculate the incident Hx */
    for (int j = idx; j < ny-1; j += stx) {
        hxi[j] += 0.5 * (ezi[j] - ezi[j+1]);
    }
}


__global__
void hfield(int nx, int ny, pmlayer *pml, float *ez, float *ihx, float *ihy, float *hx, float *hy) {
    /* calculate the Hx and Hy field */
    for (int i = idy; i < nx-1; i += sty) {
        for (int j = idx; j < ny-1; j += stx) {
            int n = i*ny+j;
            ihx[n] += ez[n] - ez[n+1];
            ihy[n] += ez[n] - ez[n+ny];
            hx[n] = pml->fy3[j] * hx[n] + pml->fy2[j] * (0.5 * ez[n] - 0.5 * ez[n+1] + pml->fx1[i] * ihx[n]);
            hy[n] = pml->fx3[i] * hy[n] - pml->fx2[i] * (0.5 * ez[n] - 0.5 * ez[n+ny] + pml->fy1[j] * ihy[n]);
        }
    }
}


__global__
void incthx(int nx, int ny, int npml, float *ezi, float *hx) {
    /* incident Hx values */
    for (int i = idx+npml-1; i <= nx-npml; i += stx) {
        hx[i*ny+(npml-2)] += 0.5 * ezi[npml-1];
        hx[i*ny+(ny-npml)] -= 0.5 * ezi[ny-npml];
    }
}


__global__
void incthy(int nx, int ny, int npml, float *ezi, float *hy) {
    /* incident Hy values */
    for (int j = idx+npml-1; j <= ny-npml; j += stx) {
        hy[(npml-2)*ny+j] -= 0.5 * ezi[j];
        hy[(nx-npml)*ny+j] += 0.5 * ezi[j];
    }
}


void pmlparam(int nx, int ny, int npml, pmlayer *pml) {
    /* calculate the two-dimensional perfectly matched layer (PML) parameters */
    for (int n = 0; n < npml; n++) {
        float xm = 0.33*(npml-n)/npml*(npml-n)/npml*(npml-n)/npml;
        float xn = 0.33*(npml-n-0.5)/npml*(npml-n-0.5)/npml*(npml-n-0.5)/npml;
        pml->fx1[n] = pml->fx1[nx-2-n] = pml->fy1[n] = pml->fy1[ny-2-n] = xn;
        pml->fx2[n] = pml->fx2[nx-2-n] = pml->fy2[n] = pml->fy2[ny-2-n] = 1/(1+xn);
        pml->gx2[n] = pml->gx2[nx-1-n] = pml->gy2[n] = pml->gy2[ny-1-n] = 1/(1+xm);
        pml->fx3[n] = pml->fx3[nx-2-n] = pml->fy3[n] = pml->fy3[ny-2-n] = (1-xn)/(1+xn);
        pml->gx3[n] = pml->gx3[nx-1-n] = pml->gy3[n] = pml->gy3[ny-1-n] = (1-xm)/(1+xm);
    }
}


int main() {

    int nx = 60;  /* number of grid points */
    int ny = 60;  /* number of grid points */

    int ns = 115;  /* number of time steps */

    float *ezi, *hxi;
    /* allocate unified memory accessible from host or device */
    hipMallocManaged(&ezi, ny*sizeof(*ezi));
    hipMallocManaged(&hxi, ny*sizeof(*hxi));

    /* initialize ezi and hxi arrays on the host */
    for (int i = 0; i < ny; i++) {
        ezi[i] = 0.0f;
        hxi[i] = 0.0f;
    }

    float *dz, *ez, *hx, *hy;
    /* allocate unified memory accessible from host or device */
    hipMallocManaged(&dz, nx*ny*sizeof(*dz));
    hipMallocManaged(&ez, nx*ny*sizeof(*ez));
    hipMallocManaged(&hx, nx*ny*sizeof(*hx));
    hipMallocManaged(&hy, nx*ny*sizeof(*hy));

    /* initialize dz, ez, hx and hy arrays on the host */
    for (int i = 0; i < nx*ny; i++) {
        dz[i] = 0.0f;
        ez[i] = 0.0f;
        hx[i] = 0.0f;
        hy[i] = 0.0f;
    }

    float *ihx, *ihy;
    /* allocate unified memory accessible from host or device */
    hipMallocManaged(&ihx, nx*ny*sizeof(*ihx));
    hipMallocManaged(&ihy, nx*ny*sizeof(*ihy));

    /* initialize ihx and ihy arrays on the host */
    for (int i = 0; i < nx*ny; i++) {
        ihx[i] = 0.0f;
        ihy[i] = 0.0f;
    }

    float *naz;
    hipMallocManaged(&naz, nx*ny*sizeof(*naz));
    for (int i = 0; i < nx*ny; naz[i] = 1.0f, i++);

    float *bc;
    hipMallocManaged(&bc, 4*sizeof(*bc));
    for (int i = 0; i < 4; bc[i] = 0.0f, i++);

    pmlayer pml;
    hipMallocManaged(&pml.fx1, nx*sizeof(*pml.fx1));
    hipMallocManaged(&pml.fx2, nx*sizeof(*pml.fx2));
    hipMallocManaged(&pml.fx3, nx*sizeof(*pml.fx3));
    hipMallocManaged(&pml.fy1, ny*sizeof(*pml.fy1));
    hipMallocManaged(&pml.fy2, ny*sizeof(*pml.fy2));
    hipMallocManaged(&pml.fy3, ny*sizeof(*pml.fy3));
    hipMallocManaged(&pml.gx2, nx*sizeof(*pml.gx2));
    hipMallocManaged(&pml.gx3, nx*sizeof(*pml.gx3));
    hipMallocManaged(&pml.gy2, ny*sizeof(*pml.gy2));
    hipMallocManaged(&pml.gy3, ny*sizeof(*pml.gy3));

    for (int i = 0; i < nx; i++) {
        pml.fx1[i] = 0.0f;
        pml.fx2[i] = 1.0f;
        pml.fx3[i] = 1.0f;
        pml.gx2[i] = 1.0f;
        pml.gx3[i] = 1.0f;
    }

    for (int i = 0; i < ny; i++) {
        pml.fy1[i] = 0.0f;
        pml.fy2[i] = 1.0f;
        pml.fy3[i] = 1.0f;
        pml.gy2[i] = 1.0f;
        pml.gy3[i] = 1.0f;
    }

    int npml = 8;  /* pml thickness */
    pmlparam(nx, ny, npml, &pml);

    float ds = 0.01;  /* spatial step (m) */
    float dt = ds/6e8;  /* time step (s) */

    dim3 gridDim, blockDim;
    blockDim.x = 16;
    blockDim.y = 16;
    gridDim.x = (ny+blockDim.x-1)/blockDim.x;
    gridDim.y = (nx+blockDim.y-1)/blockDim.y;

    for (int t = 1; t <= ns; t++) {
        ezinct<<<(ny+255)/256, 256>>>(ny, ezi, hxi, bc);
        dfield<<<gridDim, blockDim>>>(t, nx, ny, &pml, ezi, dz, hx, hy);
        inctdz<<<(nx+255)/256, 256>>>(nx, ny, npml, hxi, dz);
        efield<<<gridDim, blockDim>>>(nx, ny, naz, dz, ez);
        hxinct<<<(ny+255)/256, 256>>>(ny, ezi, hxi);
        hfield<<<gridDim, blockDim>>>(nx, ny, &pml, ez, ihx, ihy, hx, hy);
        incthx<<<(nx+255)/256, 256>>>(nx, ny, npml, ezi, hx);
        incthy<<<(ny+255)/256, 256>>>(nx, ny, npml, ezi, hy);
    }

    hipDeviceSynchronize();

    hipFree(pml.fx1);
    hipFree(pml.fx2);
    hipFree(pml.fx3);
    hipFree(pml.fy1);
    hipFree(pml.fy2);
    hipFree(pml.fy3);
    hipFree(pml.gx2);
    hipFree(pml.gx3);
    hipFree(pml.gy2);
    hipFree(pml.gy3);
    hipFree(naz);
    hipFree(ezi);
    hipFree(hxi);
    hipFree(ihx);
    hipFree(ihy);
    hipFree(bc);
    hipFree(dz);
    hipFree(ez);
    hipFree(hx);
    hipFree(hy);

    return 0;
}
