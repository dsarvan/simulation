/* File: test_2_3.cu
 * Name: D.Saravanan
 * Date: 10/01/2022
 * Simulation of a pulse striking a frequency-dependent dielectric material and
 * implements the discrete Fourier transform with a Gaussian pulse as its source
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define idx blockIdx.x*blockDim.x+threadIdx.x
#define stx blockDim.x*gridDim.x


typedef struct {
    float *nax, *nbx;
    float *ncx, *ndx;
} medium;


typedef struct {
    float *r_pt, *i_pt;
    float *r_in, *i_in;
} ftrans;


__device__
float gaussian(int t, int t0, float sigma) {
    return exp(-0.5*(t - t0)/sigma*(t - t0)/sigma);
}


__global__
void fourier(int t, int nf, int nx, float dt, float *freq, float *ex, ftrans *ft) {
    for (int n = threadIdx.y; n < nf; n += blockDim.y) {
        for (int i = idx; i < nx; i += stx) {
            /* calculate the Fourier transform of Ex field */
            int m = n*nx+i;
            ft->r_pt[m] += cos(2*M_PI*freq[n]*dt*t) * ex[i];
            ft->i_pt[m] -= sin(2*M_PI*freq[n]*dt*t) * ex[i];
        }
        if (idx == 0 && t < nx/2) {
            /* calculate the Fourier transform of input source */
            ft->r_in[n] += cos(2*M_PI*freq[n]*dt*t) * ex[10];
            ft->i_in[n] -= sin(2*M_PI*freq[n]*dt*t) * ex[10];
        }
    }
}


__global__
void dxfield(int t, int nx, float *dx, float *hy) {
    /* calculate the electric flux density Dx */
    for (int i = idx+1; i < nx; i += stx)
        dx[i] += 0.5 * (hy[i-1] - hy[i]);
    /* put a Gaussian pulse at the low end */
    if (idx == 1) dx[1] += gaussian(t, 50, 10.0f);
}


__global__
void exfield(int nx, medium *md, float *dx, float *ix, float *sx, float *ex) {
    /* calculate the Ex field from Dx */
    for (int i = idx+1; i < nx; i += stx) {
        ex[i] = md->nax[i] * (dx[i] - ix[i] - md->ncx[i] * sx[i]);
        ix[i] += md->nbx[i] * ex[i];
        sx[i] = md->ncx[i] * sx[i] + md->ndx[i] * ex[i];
    }
}


__global__
void hyfield(int nx, float *ex, float *hy, float *bc) {
    /* absorbing boundary conditions */
    if (idx == 0) ex[0] = bc[0], bc[0] = bc[1], bc[1] = ex[1];
    if (idx == nx-1) ex[nx-1] = bc[3], bc[3] = bc[2], bc[2] = ex[nx-2];
    /* calculate the Hy field */
    for (int i = idx; i < nx-1; i += stx)
        hy[i] += 0.5 * (ex[i] - ex[i+1]);
}


medium dielectric(int nx, float dt, float chi, float tau, float epsr, float sigma) {
    medium md;
    hipMallocManaged(&md.nax, nx*sizeof(*md.nax));
    hipMallocManaged(&md.nbx, nx*sizeof(*md.nbx));
    hipMallocManaged(&md.ncx, nx*sizeof(*md.ncx));
    hipMallocManaged(&md.ndx, nx*sizeof(*md.ndx));
    for (int i = 0; i < nx; md.nax[i] = 1.0f, i++);
    for (int i = 0; i < nx; md.nbx[i] = 0.0f, i++);
    for (int i = 0; i < nx; md.ncx[i] = 0.0f, i++);
    for (int i = 0; i < nx; md.ndx[i] = 0.0f, i++);
    float eps0 = 8.854e-12;  /* vacuum permittivity (F/m) */
    for (int i = nx/2; i < nx; i++) {
        md.nax[i] = 1/(epsr + sigma*dt/eps0 + chi*dt/tau);
        md.nbx[i] = sigma*dt/eps0;
        md.ncx[i] = exp(-dt/tau);
        md.ndx[i] = chi*dt/tau;
    }
    return md;
}


int main() {

    int nx = 38000;  /* number of grid points */
    int ns = 40000;  /* number of time steps */

    float *dx, *ex, *ix, *sx, *hy;
    /* allocate unified memory accessible from host or device */
    hipMallocManaged(&dx, nx*sizeof(*dx));
    hipMallocManaged(&ex, nx*sizeof(*ex));
    hipMallocManaged(&ix, nx*sizeof(*ix));
    hipMallocManaged(&sx, nx*sizeof(*sx));
    hipMallocManaged(&hy, nx*sizeof(*hy));

    /* initialize dx, ex, ix, sx and hy arrays on the host */
    for (int i = 0; i < nx; i++) {
        dx[i] = 0.0f;
        ex[i] = 0.0f;
        ix[i] = 0.0f;
        sx[i] = 0.0f;
        hy[i] = 0.0f;
    }

    float *bc;
    hipMallocManaged(&bc, 4*sizeof(*bc));
    for (int i = 0; i < 4; bc[i] = 0.0f, i++);

    float ds = 0.01;  /* spatial step (m) */
    float dt = ds/6e8;  /* time step (s) */
    float chi = 2.0;  /* relaxation susceptibility */
    float tau = 0.001e-6;  /* relaxation time (s) */
    float epsr = 2.0;  /* relative permittivity */
    float sigma = 0.01;  /* conductivity (S/m) */
    medium md = dielectric(nx, dt, chi, tau, epsr, sigma);

    int nf = 3;  /* number of frequencies */
    /* frequency 50 MHz, 200 MHz, 500 MHz */
    float *freq;
    hipMallocManaged(&freq, nf*sizeof(*freq));
    freq[0] = 50e6; freq[1] = 200e6; freq[2] = 500e6;

    ftrans ft;
    hipMallocManaged(&ft.r_pt, nf*nx*sizeof(*ft.r_pt));
    hipMallocManaged(&ft.i_pt, nf*nx*sizeof(*ft.i_pt));
    hipMallocManaged(&ft.r_in, nf*sizeof(*ft.r_in));
    hipMallocManaged(&ft.i_in, nf*sizeof(*ft.i_in));

    for (int i = 0; i < nf*nx; i++) {
        ft.r_pt[i] = 0.0f;
        ft.i_pt[i] = 0.0f;
    }

    for (int i = 0; i < nf; i++) {
        ft.r_in[i] = 0.0f;
        ft.i_in[i] = 0.0f;
    }

    float *amplt, *phase;
    hipMallocManaged(&amplt, nf*nx*sizeof(*amplt));
    hipMallocManaged(&phase, nf*nx*sizeof(*phase));

    for (int i = 0; i < nf*nx; i++) {
        amplt[i] = 0.0f;
        phase[i] = 0.0f;
    }

    dim3 gridDim, blockDim;
    blockDim.x = 256;
    gridDim.x = (nx+blockDim.x-1)/blockDim.x;

    hipEvent_t stime, ntime;
    hipEventCreate(&stime);
    hipEventCreate(&ntime);

    hipEventRecord(stime, 0);

    for (int t = 1; t <= ns; t++) {
        dxfield<<<gridDim, blockDim>>>(t, nx, dx, hy);
        exfield<<<gridDim, blockDim>>>(nx, &md, dx, ix, sx, ex);
        fourier<<<gridDim, dim3(256,4)>>>(t, nf, nx, dt, freq, ex, &ft);
        hyfield<<<gridDim, blockDim>>>(nx, ex, hy, bc);
    }

    hipDeviceSynchronize();

    /* calculate the amplitude and phase at each frequency */
    for (int n = 0; n < nf; n++) {
        for (int i = 0; i < nx; i++) {
            int m = n*nx+i;
            amplt[m] = 1/hypotf(ft.r_in[n],ft.i_in[n]) * hypotf(ft.r_pt[m],ft.i_pt[m]);
            phase[m] = atan2f(ft.i_pt[m],ft.r_pt[m]) - atan2f(ft.i_in[n],ft.r_in[n]);
        }
    }

    hipEventRecord(ntime, 0);
    hipEventSynchronize(ntime);

    float time;
    hipEventElapsedTime(&time, stime, ntime);
    printf("Total compute time on GPU: %.3f s\n", time/1000.0f);

    hipEventDestroy(stime);
    hipEventDestroy(ntime);

    for (int i = 0; i < 50; i++)
        printf("%e\n", ex[i]);

    hipFree(ft.r_pt);
    hipFree(ft.i_pt);
    hipFree(ft.r_in);
    hipFree(ft.i_in);
    hipFree(md.nax);
    hipFree(md.nbx);
    hipFree(md.ncx);
    hipFree(md.ndx);
    hipFree(amplt);
    hipFree(phase);
    hipFree(freq);
    hipFree(bc);
    hipFree(dx);
    hipFree(ex);
    hipFree(ix);
    hipFree(sx);
    hipFree(hy);

    return 0;
}
